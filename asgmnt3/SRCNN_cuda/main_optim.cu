#include <iostream>
#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>

#include <sstream>
#include <fstream>
using namespace std;

typedef float d_type;

void _padding(
    d_type *in, d_type *in_padded,
    int N,
    int C, int H_in, int W_in,
    int padding
) {
    const int H_pad = H_in + padding * 2;
    const int W_pad = W_in + padding * 2;
    // const int MAX_pad = N * C * H_pad * W_pad;
    
    // calloc has already set all in_padded to zero
    int n, c, h, w, i = 0;
    for (n=0; n < N; n++) {
        for (c=0; c < C; c++) {
            for (h=0; h < H_in; h++) {
                for (w=0; w < W_in; w++) {
                    // in_padded[n][c][h+padding][w+padding] = in[n][c][h][w]
                    in_padded[n*(C*H_pad*W_pad) + c*(H_pad*W_pad) + (h+padding)*(W_pad) + (w+padding)] = in[i++];
                }
            }
        }
    }
}

__global__ void convolution_kernel(float *input, float *kernel, float *output,
                                    int N, int C, int H_in, int W_in,
                                    int F, int H_w, int W_w, int H_out, int W_out,
                                    int stride, int H_pad, int W_pad)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N*F*H_out*W_out) {
        int w = idx % W_out;
        int h = (idx / W_out) % H_out;
        int f = (idx / (H_out*W_out)) % F;
        int n = idx / (F*H_out*W_out);

        float sum = 0.0;
        for (int c = 0; c < C; c++) {
            for (int hh = 0; hh < H_w; hh++) {
                for (int ww = 0; ww < W_w; ww++) {
                    int h_in = h * stride + hh - H_pad;
                    int w_in = w * stride + ww - W_pad;
                    if (h_in >= 0 && h_in < H_in && w_in >= 0 && w_in < W_in) {
                        int input_idx = n*C*H_in*W_in + c*H_in*W_in + h_in*W_in + w_in;
                        int kernel_idx = f*C*H_w*W_w + c*H_w*W_w + hh*W_w + ww;
                        sum += input[input_idx] * kernel[kernel_idx];
                    }
                }
            }
        }
        int output_idx = n*F*H_out*W_out + f*H_out*W_out + h*W_out + w;
        output[output_idx] = sum;
    }
}

__global__ void addition_kernel(
    float* buffer,
    const float* conv_bias,
    int f_len, int h_len, int w_len
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < f_len * h_len * w_len; i += stride) {
        int n = 0, f = i / (h_len * w_len), h = (i / w_len) % h_len, w = i % w_len;
        buffer[n * f_len * h_len * w_len + f * h_len * w_len + h * w_len + w] += conv_bias[f];
    }
}

__global__ void padding_kernel(d_type *in, d_type *in_padded, int C, int H_in, int W_in, int H_pad, int W_pad, int padding) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n = idx / (C * H_in * W_in);
    int c = (idx / (H_in * W_in)) % C;
    int h = (idx / W_in) % H_in;
    int w = idx % W_in;
    if (idx < C * H_in * W_in) {
        in_padded[n * (C * H_pad * W_pad) + c * (H_pad * W_pad) + (h + padding) * W_pad + (w + padding)] = in[n * (C * H_in * W_in) + c * (H_in * W_in) + h * W_in + w];
    }
}

__global__ void relu_kernel(
    float* buffer,
    int f_len, int h_len, int w_len
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < f_len * h_len * w_len; i += stride) {
        int n = 0, f = i / (h_len * w_len), h = (i / w_len) % h_len, w = i % w_len;
        buffer[n * f_len * h_len * w_len + f * h_len * w_len + h * w_len + w] = buffer[n * f_len * h_len * w_len + f * h_len * w_len + h * w_len + w] > 0.0 ? buffer[n * f_len * h_len * w_len + f * h_len * w_len + h * w_len + w] : 0.0;
    }
}

int conv (float *input, float *kernel, float *output,
            int N, int C, int H_in, int W_in,
            int F, int H_w, int W_w, int H_out, int W_out,
            int stride, int H_pad, int W_pad)
{
    int num_blocks = (N*F*H_out*W_out + 255) / 256;
    int num_threads = 256;
    convolution_kernel<<<num_blocks, num_threads>>>(input, kernel, output, 
                                                    N, C, H_in, W_in, 
                                                    F, H_w, W_w, H_out, W_out, 
                                                    stride, H_pad, W_pad);

    // cudaError_t cudaStatus;

    // cudaStatus = cudaGetLastError();
    // if (cudaStatus != cudaSuccess) {
    //     printf("convolution_kernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
    //     return 1;
    // }

    // cudaStatus = cudaDeviceSynchronize();
    // if (cudaStatus != cudaSuccess) {
    //     printf("cudaDeviceSynchronize returned error code %d after launching convolution_kernel!\n", cudaStatus);
    //     return 1;
    // }

    return 0;
}

int conv_bias_add (
    float* buffer,
    const float* conv_bias,
    int F, int H, int W
) {
    int block_size = 256;
    int grid_size = (1 * F * H * W + block_size - 1) / block_size;
    addition_kernel<<<grid_size, block_size>>>(buffer, conv_bias, F, H, W);
    
    // cudaError_t cudaStatus;

    // cudaStatus = cudaGetLastError();
    // if (cudaStatus != cudaSuccess) {
    //     printf("addition_kernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
    //     return 1;
    // }

    // cudaStatus = cudaDeviceSynchronize();
    // if (cudaStatus != cudaSuccess) {
    //     printf("cudaDeviceSynchronize returned error code %d after launching addition_kernel!\n", cudaStatus);
    //     return 1;
    // }

    return 0;
}


int padding(d_type *in, d_type *in_padded, int N, int C, int H_in, int W_in, int padding) {
    const int H_pad = H_in + padding * 2;
    const int W_pad = W_in + padding * 2;
    int num_threads = N * C * H_in * W_in;
    padding_kernel<<<(num_threads + 255) / 256, 256>>>(in, in_padded, C, H_in, W_in, H_pad, W_pad, padding);

    // cudaError_t cudaStatus;

    // cudaStatus = cudaGetLastError();
    // if (cudaStatus != cudaSuccess) {
    //     printf("addition_kernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
    //     return 1;
    // }

    // cudaStatus = cudaDeviceSynchronize();
    // if (cudaStatus != cudaSuccess) {
    //     printf("cudaDeviceSynchronize returned error code %d after launching addition_kernel!\n", cudaStatus);
    //     return 1;
    // }

    return 0;
}

int relu (
    float* buffer,
    int F, int H, int W
) {
    int block_size = 256;
    int grid_size = (1 * F * H * W + block_size - 1) / block_size;
    relu_kernel<<<grid_size, block_size>>>(buffer, F, H, W);
    
    // cudaError_t cudaStatus;

    // cudaStatus = cudaGetLastError();
    // if (cudaStatus != cudaSuccess) {
    //     printf("addition_kernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
    //     return 1;
    // }

    // cudaStatus = cudaDeviceSynchronize();
    // if (cudaStatus != cudaSuccess) {
    //     printf("cudaDeviceSynchronize returned error code %d after launching addition_kernel!\n", cudaStatus);
    //     return 1;
    // }

    return 0;
}


void CONV_forward_pass(
    d_type *in, d_type *out, d_type *weight, d_type *bias,
    int N,
    int C, int H_in, int W_in,
    int F, int H_w, int W_w,
    int H_out, int W_out,
    int stride, int padding
) {
    // pad input
    const int H_pad = H_in + padding * 2;
    const int W_pad = W_in + padding * 2;

    d_type *in_padded = (d_type *)calloc(N * C * H_pad * W_pad, sizeof(d_type));
    _padding(in, in_padded, N, C, H_in, W_in, padding);

    // conv operation
    int n, f, h, w, c, hh, ww, i=0;
    for (n=0; n < N; n++) {
        for (f=0; f < F; f++) {
            for (h=0; h < H_out; h++) {
                for (w=0; w < W_out; w++) {
                    
                    out[i] = bias[f];
                    for (c=0; c < C; c++) {
                        for (hh=0; hh < H_w; hh++) {
                            for (ww=0; ww < W_w; ww++) {
                                // out[n, f, h, w] += weight[f, c, hh, ww] * in_padded[n, c, h*stride+hh, w*stride+ww]
                                out[i] += weight[f*(C*H_w*W_w) + c*(H_w*W_w) + hh*(W_w) + ww] * in_padded[n*(C*H_pad*W_pad) + c*(H_pad*W_pad) + (h*stride+hh)*(W_pad) + (w*stride+ww)];
                            }
                        }
                    }
                    ++i;

                }
            }
        }
    }
    free(in_padded);
}

void RELU_forward_pass(
    d_type *in, d_type *out,
    int N, int C, int H, int W
) {
    // Relu::in = in;
    const int num = N*C*H*W;
    int i=0;
    
    while (i < num) {
        out[i] = in[i] > 0 ? in[i] : 0;
        ++i;
    }
}

void read_params(string file_path, d_type *param) {
    ifstream csvread;
    csvread.open(file_path, ios::in);
    if(csvread) {
        string s;
        int i=0;
        while(getline(csvread, s, '\n')) {
            param[i++] = stof(s);
        }
        csvread.close();
    }
    else{
        cerr << "Unable to read parameter!" << endl;
    exit (EXIT_FAILURE);
    }
}

void save_img(string file_path, d_type *param, const int length) {
    ofstream outfile;
	outfile.open(file_path);
    for (int i=0; i < length; i++) {
        outfile << param[i] <<endl;
    }
	outfile.close();
}

int main() {
    d_type *conv1_weight = (d_type *)calloc(64*3*9*9, sizeof(d_type));
    d_type *conv1_bias = (d_type *)calloc(64, sizeof(d_type));
    d_type *conv2_weight = (d_type *)calloc(64*32*1*1, sizeof(d_type));
    d_type *conv2_bias = (d_type *)calloc(32, sizeof(d_type));
    d_type *conv3_weight = (d_type *)calloc(32*3*5*5, sizeof(d_type));
    d_type *conv3_bias = (d_type *)calloc(3, sizeof(d_type));

    read_params("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/params/conv1_weight.csv", conv1_weight);
    read_params("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/params/conv1_bias.csv", conv1_bias);
    read_params("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/params/conv2_weight.csv", conv2_weight);
    read_params("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/params/conv2_bias.csv", conv2_bias);
    read_params("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/params/conv3_weight.csv", conv3_weight);
    read_params("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/params/conv3_bias.csv", conv3_bias);

    d_type *img = (d_type *)calloc(3*288*352, sizeof(d_type));
    d_type *res_img = (d_type *)calloc(3*288*352, sizeof(d_type));

    read_params("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/input_image/img0.csv", img);

    d_type *temp_buff = (d_type *)calloc(1*64*284*348, sizeof(d_type));
    d_type *temp_buff2 = (d_type *)calloc(1*32*288*352, sizeof(d_type));



    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int inputWidth = 352;
    int inputHeight = 288;
    int inputDepth = 3;

    // allocate memory on device for input, output, and kernel data
    float *d_input, *d_pad_input, *d_output, *d_conv1_weight, *d_conv1_bias, *d_conv2_weight, *d_conv2_bias, *d_conv3_weight, *d_conv3_bias;
    hipMalloc((void **)&d_input, inputWidth * inputHeight * inputDepth * sizeof(float));
    hipMalloc((void **)&d_pad_input, 1*64*288*352 * sizeof(float));
    hipMalloc((void **)&d_output, 1*64*284*348 * sizeof(float));
    hipMalloc((void **)&d_conv1_weight, 64*3*9*9 * sizeof(float));
    hipMalloc((void **)&d_conv1_bias, 64 * sizeof(float));
    hipMalloc((void **)&d_conv2_weight, 32*64*1*1 * sizeof(float));
    hipMalloc((void **)&d_conv2_bias, 32 * sizeof(float));
    hipMalloc((void **)&d_conv3_weight, 3*32*5*5 * sizeof(float));
    hipMalloc((void **)&d_conv3_bias, 3 * sizeof(float));


    // copy input and kernel data from host to device
    hipMemcpy(d_input, img, inputWidth * inputHeight * inputDepth * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_weight, conv1_weight, 64*3*9*9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_bias, conv1_bias, 64 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_weight, conv2_weight, 32*64*1*1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_bias, conv2_bias, 32 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_weight, conv3_weight, 3*32*5*5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_bias, conv3_bias, 3 * sizeof(float), hipMemcpyHostToDevice);

    auto start_time = std::chrono::high_resolution_clock::now();

    // CONV_forward_pass(img, temp_buff, conv1_weight, conv1_bias,
    //                     1, 3, 288, 352, 64, 9, 9, 284, 348, 1, 2);
    // RELU_forward_pass(temp_buff, temp_buff,
    //                     1, 64, 284, 348);
    padding(d_input, d_pad_input, 1, 3, 288, 352, 2);
    conv(d_pad_input, d_conv1_weight, d_output,
                        1, 3, 292, 356, 64, 9, 9, 284, 348, 1, 2, 2);
    conv_bias_add(d_output, d_conv1_bias, 
                        64, 284, 348);
    relu(d_output, 64, 284, 348);


    // CONV_forward_pass(temp_buff, temp_buff2, conv2_weight, conv2_bias,
    //                     1, 64, 284, 348, 32, 1, 1, 288, 352, 1, 2);
    // RELU_forward_pass(temp_buff2, temp_buff2,
    //                     1, 32, 288, 352);
    padding(d_output, d_pad_input, 1, 64, 284, 348, 2);
    conv(d_pad_input, d_conv2_weight, d_output,
                        1, 64, 288, 352, 32, 1, 1, 288, 352, 1, 2, 2);
    conv_bias_add(d_output, d_conv2_bias, 
                        32, 288, 352);
    relu(d_output, 32, 288, 352);


    // CONV_forward_pass(temp_buff2, res_img, conv3_weight, conv3_bias,
    //                     1, 32, 288, 352, 3, 5, 5, 288, 352, 1, 2);
    padding(d_output, d_pad_input, 1, 32, 288, 352, 2);
    conv(d_pad_input, d_conv3_weight, d_output,
                        1, 32, 292, 356, 3, 5, 5, 288, 352, 1, 2, 2);
    conv_bias_add(d_output, d_conv3_bias, 
                        3, 288, 352);

    // cudaStatus = cudaDeviceSynchronize();
    
    // copy output data from device to host
    hipMemcpy(res_img, d_output, 1*3*288*352 * sizeof(float), hipMemcpyDeviceToHost);

    auto end_time = std::chrono::high_resolution_clock::now();
	auto duration_us = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
    std::cout << "Finished inferencing (manual forward pass) in: " << duration_us / 1000.0 << " ms" << std::endl;

    // free memory on device
    hipFree(d_input);
    hipFree(d_pad_input);
    hipFree(d_output);
    hipFree(d_conv1_weight);
    hipFree(d_conv1_bias);
    hipFree(d_conv2_weight);
    hipFree(d_conv2_bias);
    hipFree(d_conv3_weight);
    hipFree(d_conv3_bias);


    save_img("/home/chengyih001/Documents/ECE1718/A3/asgmnt3_local/SRCNN_cpp_simple/output_image/img0_res.csv", res_img, 3*288*352);

    free(conv1_weight);
    free(conv1_bias);
    free(conv2_weight);
    free(conv2_bias);
    free(conv3_weight);
    free(conv3_bias);

    free(img);
    free(res_img);
    free(temp_buff);
    free(temp_buff2);
}